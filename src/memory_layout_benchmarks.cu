#include "hip/hip_runtime.h"
#include "memory_layout_benchmarks.cuh"
#include "bignum_types.h"
#include "bignum_conversions.h"
#include "constants.h"
#include <stdint.h>

void benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, uint32_t threads_per_block, uint32_t blocks_per_grid)
{
    // arrange data in coalesced form
    bignum_array_to_coalesced_bignum_array(host_a);
    bignum_array_to_coalesced_bignum_array(host_b);
    bignum_array_to_coalesced_bignum_array(host_c);

    // device operands (dev_a, dev_b) and results (dev_c)
    uint32_t* dev_a;
    uint32_t* dev_b;
    uint32_t* dev_c;

    // allocate gpu memory
    hipMalloc((void**) &dev_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipMalloc((void**) &dev_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipMalloc((void**) &dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));

    // copy operands to device memory
    hipMemcpy(dev_a, host_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);

    // execute benchmark kernels
    kernel_1<<<blocks_per_grid, threads_per_block>>>(dev_c, dev_a, dev_b);

    // copy results back to host
    hipMemcpy(host_c, dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // put data back to non-coalesced form
    coalesced_bignum_array_to_bignum_array(host_a);
    coalesced_bignum_array_to_bignum_array(host_b);
    coalesced_bignum_array_to_bignum_array(host_c);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

__global__ void kernel_1(uint32_t* c, uint32_t* a, uint32_t* b)
{
    // add(c, a, b);

    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    while (tid < NUMBER_OF_BIGNUMS)
    {
        asm("add.cc.u32 %0, %1, %2;"
            : "=r"(c[COAL_IDX(0, tid)])
            : "r" (a[COAL_IDX(0, tid)]),
              "r" (b[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("addc.cc.u32 %0, %1, %2;"
                : "=r"(c[COAL_IDX(i, tid)])
                : "r" (a[COAL_IDX(i, tid)]),
                  "r" (b[COAL_IDX(i, tid)]));
        }

        asm("addc.u32 %0, %1, %2;"
            : "=r"(c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
              "r" (b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        tid += stride;
    }
}

// __device__ void add(uint32_t* c, uint32_t* a, uint32_t* b)
// {
//     uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
//     uint32_t stride = blockDim.x * gridDim.x;

//     while (tid < NUMBER_OF_BIGNUMS)
//     {
//         asm("add.cc.u32 %0, %1, %2;"
//             : "=r"(c[COAL_IDX(0, tid)])
//             : "r" (a[COAL_IDX(0, tid)]),
//               "r" (b[COAL_IDX(0, tid)]));

//         #pragma unroll
//         for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
//         {
//             asm("addc.cc.u32 %0, %1, %2;"
//                 : "=r"(c[COAL_IDX(i, tid)])
//                 : "r" (a[COAL_IDX(i, tid)]),
//                   "r" (b[COAL_IDX(i, tid)]));
//         }

//         asm("addc.u32 %0, %1, %2;"
//             : "=r"(c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
//             : "r" (a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
//               "r" (b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

//         tid += stride;
//     }
// }

// __device__ void subtract(uint32_t* c, uint32_t* a, uint32_t* b)
// {
//     uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
//     uint32_t stride = blockDim.x * gridDim.x;

//     while (tid < NUMBER_OF_BIGNUMS)
//     {
//         asm("sub.cc.u32 %0, %1, %2;"
//             : "=r"(c[COAL_IDX(0, tid)])
//             : "r" (a[COAL_IDX(0, tid)]),
//               "r" (b[COAL_IDX(0, tid)]));

//         #pragma unroll
//         for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
//         {
//             asm("subc.cc.u32 %0, %1, %2;"
//                 : "=r"(c[COAL_IDX(i, tid)])
//                 : "r" (a[COAL_IDX(i, tid)]),
//                   "r" (b[COAL_IDX(i, tid)]));
//         }

//         asm("subc.u32 %0, %1, %2;"
//             : "=r"(c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
//             : "r" (a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
//               "r" (b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

//         tid += stride;
//     }
// }
