#include "hip/hip_runtime.h"
#include "interleaved_addition_benchmark.cuh"
#include "test_constants.h"
#include "bignum_conversions.h"

#include <gmp.h>

void execute_interleaved_addition_on_device(bignum* host_c, bignum* host_a,
                                            bignum* host_b,
                                            uint32_t threads_per_block,
                                            uint32_t blocks_per_grid)
{
    // for this interleaved addition, we are going to interleave the values of
    // the 2 operands host_a and host_b.
    // Our operands will look like the following:

    // host_a[0][0], host_b[0][0], host_a[0][1], host_b[0][1],
    // host_a[0][2], host_b[0][2], host_a[0][3], host_b[0][3],
    // host_a[0][4], host_b[0][4], host_a[1][0], host_b[1][0], ...

    // our results will be stocked sequentially as for normal addition.

    interleaved_bignum* host_interleaved_operands =
        (interleaved_bignum*) calloc(NUMBER_OF_TESTS, sizeof(interleaved_bignum));

    // interleave values of host_a and host_b in host_interleaved_operands.
    for (uint32_t i = 0; i < NUMBER_OF_TESTS; i++)
    {
        for (uint32_t j = 0; j < INTERLEAVED_BIGNUM_NUMBER_OF_WORDS; j++)
        {
            if (j % 2 == 0)
            {
                host_interleaved_operands[i][j] = host_a[i][j / 2];
            }
            else
            {
                host_interleaved_operands[i][j] = host_b[i][j / 2];
            }
        }
    }

    // device operands (dev_interleaved_operands) and results (dev_results)
    interleaved_bignum* dev_interleaved_operands;
    bignum* dev_results;

    hipMalloc((void**) &dev_interleaved_operands,
               NUMBER_OF_TESTS * sizeof(interleaved_bignum));
    hipMalloc((void**) &dev_results, NUMBER_OF_TESTS * sizeof(bignum));

    // copy operands to device memory
    hipMemcpy(dev_interleaved_operands, host_interleaved_operands,
               NUMBER_OF_TESTS * sizeof(interleaved_bignum),
               hipMemcpyHostToDevice);

    // free host_interleaved_operands which we no longer need.
    free(host_interleaved_operands);

    interleaved_addition<<<blocks_per_grid, threads_per_block>>>(
        dev_results, dev_interleaved_operands);

    // copy results back to host
    hipMemcpy(host_c, dev_results, NUMBER_OF_TESTS * sizeof(bignum),
               hipMemcpyDeviceToHost);

    // free device memory
    hipFree(dev_interleaved_operands);
    hipFree(dev_results);
}

__global__ void interleaved_addition(bignum* dev_results,
                                     interleaved_bignum* dev_interleaved_operands)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (tid < NUMBER_OF_TESTS)
    {
        asm("add.cc.u32  %0, %1, %2;"
            : "=r"(dev_results[tid][0])
            : "r"(dev_interleaved_operands[tid][0]),
              "r"(dev_interleaved_operands[tid][1])
            );

        asm("addc.cc.u32 %0, %1, %2;"
            : "=r"(dev_results[tid][1])
            : "r"(dev_interleaved_operands[tid][2]),
              "r"(dev_interleaved_operands[tid][3])
            );

        asm("addc.cc.u32 %0, %1, %2;"
            : "=r"(dev_results[tid][2])
            : "r"(dev_interleaved_operands[tid][4]),
              "r"(dev_interleaved_operands[tid][5])
            );

        asm("addc.cc.u32 %0, %1, %2;"
            : "=r"(dev_results[tid][3])
            : "r"(dev_interleaved_operands[tid][6]),
              "r"(dev_interleaved_operands[tid][7])
            );

        asm("addc.u32    %0, %1, %2;"
            : "=r"(dev_results[tid][4])
            : "r"(dev_interleaved_operands[tid][8]),
              "r"(dev_interleaved_operands[tid][9])
            );

        tid += blockDim.x * gridDim.x;
    }
}

/**
 * Checks if host_a op host_b == host_c, where host_c is to be tested against
 * values computed by gmp. If you have data in any other formats than these, you
 * will have to "rearrange" them to meet this pattern for the check to work.
 * @param host_c Values we have computed with our algorithms.
 * @param host_a First operands.
 * @param host_b Second operands.
 */
void check_interleaved_addition_results(bignum* host_c, bignum* host_a,
                                        bignum* host_b)
{
    bool results_correct = true;

    for (uint32_t i = 0; results_correct && i < NUMBER_OF_TESTS; i++)
    {
        char* bignum_a_str = bignum_to_string(host_a[i]);
        char* bignum_b_str = bignum_to_string(host_b[i]);
        char* bignum_c_str = bignum_to_string(host_c[i]);

        mpz_t gmp_bignum_a;
        mpz_t gmp_bignum_b;
        mpz_t gmp_bignum_c;

        mpz_init_set_str(gmp_bignum_a, bignum_a_str, 2);
        mpz_init_set_str(gmp_bignum_b, bignum_b_str, 2);
        mpz_init(gmp_bignum_c);

        // GMP function which will calculate what our algorithm is supposed to
        // calculate
        mpz_add(gmp_bignum_c, gmp_bignum_a, gmp_bignum_b);

        // get binary string result
        char* gmp_bignum_c_str = mpz_get_str(NULL, 2, gmp_bignum_c);
        pad_string_with_zeros(&gmp_bignum_c_str);

        if (strcmp(gmp_bignum_c_str, bignum_c_str) != 0)
        {
            printf("incorrect calculation at iteration %d\n", i);
            results_correct = false;
            printf("own\n%s +\n%s =\n%s\n", bignum_a_str, bignum_b_str,
                   bignum_c_str);
            printf("gmp\n%s +\n%s =\n%s\n", bignum_a_str, bignum_b_str,
                   gmp_bignum_c_str);
        }

        free(bignum_a_str);
        free(bignum_b_str);
        free(bignum_c_str);
        free(gmp_bignum_c_str);

        mpz_clear(gmp_bignum_a);
        mpz_clear(gmp_bignum_b);
        mpz_clear(gmp_bignum_c);
    }

    if (results_correct)
    {
        printf("all correct\n");
    }
    else
    {
        printf("something wrong\n");
    }
}
