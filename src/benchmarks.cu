#include "hip/hip_runtime.h"
#include "benchmarks.h"
#include "bignum_types.h"
#include "input_output.h"
#include "operations.h"
#include "constants.h"
#include <stdio.h>
#include <stdint.h>
#include <assert.h>

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), char* operation_name);

void add_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, const char* output_file_name);
void sub_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, const char* output_file_name);
void mul_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, const char* output_file_name);

__global__ void add_glo_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__global__ void add_loc_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__global__ void sub_glo_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__global__ void sub_loc_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__global__ void mul_glo_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__global__ void mul_loc_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// BENCHMARKS /////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void add_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, const char* output_file_name)
{
    binary_operator_benchmark(host_c, host_a, host_b, add_glo_kernel, "add_glo");
    binary_operator_benchmark(host_c, host_a, host_b, add_loc_kernel, "add_loc");

    write_coalesced_bignums_to_file(output_file_name, host_c);
}

void sub_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, const char* output_file_name)
{
    binary_operator_benchmark(host_c, host_a, host_b, sub_glo_kernel, "sub_glo");
    binary_operator_benchmark(host_c, host_a, host_b, sub_loc_kernel, "sub_loc");

    write_coalesced_bignums_to_file(output_file_name, host_c);
}

void mul_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, const char* output_file_name)
{
    binary_operator_benchmark(host_c, host_a, host_b, mul_glo_kernel, "mul_glo");
    binary_operator_benchmark(host_c, host_a, host_b, mul_loc_kernel, "mul_loc");

    write_coalesced_bignums_to_file(output_file_name, host_c);
}

////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////// KERNELS ///////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

__global__ void add_glo_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    // #pragma unroll
    for (uint32_t i = 0; i < BENCHMARK_ITERATIONS; i++)
    {
        add_glo(dev_c, dev_a, dev_b, tid);
    }
}

__global__ void add_loc_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t a[MAX_BIGNUM_NUMBER_OF_WORDS];
    uint32_t b[MAX_BIGNUM_NUMBER_OF_WORDS];
    uint32_t c[MAX_BIGNUM_NUMBER_OF_WORDS];

    // #pragma unroll
    for (uint32_t i = 0; i < MAX_BIGNUM_NUMBER_OF_WORDS; i++)
    {
        a[i] = dev_a[COAL_IDX(i, tid)];
        b[i] = dev_b[COAL_IDX(i, tid)];
    }

    // #pragma unroll
    for (uint32_t i = 0; i < BENCHMARK_ITERATIONS; i++)
    {
        add_loc(c, a, b);
    }

    // #pragma unroll
    for (uint32_t i = 0; i < MAX_BIGNUM_NUMBER_OF_WORDS; i++)
    {
        dev_c[COAL_IDX(i, tid)] = c[i];
    }
}

__global__ void sub_glo_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    // #pragma unroll
    for (uint32_t i = 0; i < BENCHMARK_ITERATIONS; i++)
    {
        sub_glo(dev_c, dev_a, dev_b, tid);
    }
}

__global__ void sub_loc_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t a[MAX_BIGNUM_NUMBER_OF_WORDS];
    uint32_t b[MAX_BIGNUM_NUMBER_OF_WORDS];
    uint32_t c[MAX_BIGNUM_NUMBER_OF_WORDS];

    // #pragma unroll
    for (uint32_t i = 0; i < MAX_BIGNUM_NUMBER_OF_WORDS; i++)
    {
        a[i] = dev_a[COAL_IDX(i, tid)];
        b[i] = dev_b[COAL_IDX(i, tid)];
    }

    // #pragma unroll
    for (uint32_t i = 0; i < BENCHMARK_ITERATIONS; i++)
    {
        sub_loc(c, a, b);
    }

    // #pragma unroll
    for (uint32_t i = 0; i < MAX_BIGNUM_NUMBER_OF_WORDS; i++)
    {
        dev_c[COAL_IDX(i, tid)] = c[i];
    }
}

__global__ void mul_glo_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    // #pragma unroll
    for (uint32_t i = 0; i < BENCHMARK_ITERATIONS; i++)
    {
        mul_glo(dev_c, dev_a, dev_b, tid);
    }
}

__global__ void mul_loc_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t a[MAX_BIGNUM_NUMBER_OF_WORDS];
    uint32_t b[MAX_BIGNUM_NUMBER_OF_WORDS];
    uint32_t c[MAX_BIGNUM_NUMBER_OF_WORDS];

    // #pragma unroll
    for (uint32_t i = 0; i < MAX_BIGNUM_NUMBER_OF_WORDS; i++)
    {
        a[i] = dev_a[COAL_IDX(i, tid)];
        b[i] = dev_b[COAL_IDX(i, tid)];
    }

    // #pragma unroll
    for (uint32_t i = 0; i < BENCHMARK_ITERATIONS; i++)
    {
        mul_loc(c, a, b);
    }

    // #pragma unroll
    for (uint32_t i = 0; i < MAX_BIGNUM_NUMBER_OF_WORDS; i++)
    {
        dev_c[COAL_IDX(i, tid)] = c[i];
    }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////// GENERIC LAUNCH CONFIGURATIONS ///////////////////////
////////////////////////////////////////////////////////////////////////////////

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), char* operation_name)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);
    assert(kernel != NULL);
    assert(operation_name != NULL);

    // device operands (dev_a, dev_b) and results (dev_c)
    uint32_t* dev_a;
    uint32_t* dev_b;
    uint32_t* dev_c;

    // allocate gpu memory
    hipError_t dev_a_malloc_success = hipMalloc((void**) &dev_a, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_b_malloc_success = hipMalloc((void**) &dev_b, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_c_malloc_success = hipMalloc((void**) &dev_c, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    assert(dev_a_malloc_success == hipSuccess);
    assert(dev_b_malloc_success == hipSuccess);
    assert(dev_c_malloc_success == hipSuccess);

    // make sure gpu memory is clean before our calculations (you never know ...)
    hipError_t dev_a_cleanup_memset_success = hipMemset(dev_a, 0, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_b_cleanup_memset_success = hipMemset(dev_b, 0, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_c_cleanup_memset_success = hipMemset(dev_c, 0, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    assert(dev_a_cleanup_memset_success == hipSuccess);
    assert(dev_b_cleanup_memset_success == hipSuccess);
    assert(dev_c_cleanup_memset_success == hipSuccess);

    // copy operands to device memory
    hipError_t dev_a_memcpy_succes = hipMemcpy(dev_a, host_a, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipError_t dev_b_memcpy_succes = hipMemcpy(dev_b, host_b, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);
    assert(dev_a_memcpy_succes == hipSuccess);
    assert(dev_b_memcpy_succes == hipSuccess);

    printf("Benchmarking \"%s\" on GPU ... ", operation_name);
    fflush(stdout);

    // execute kernel
    kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dev_c, dev_a, dev_b);

    printf("done\n");
    fflush(stdout);

    // copy results back to host
    hipError_t dev_c_memcpy_success = hipMemcpy(host_c, dev_c, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyDeviceToHost);
    assert(dev_c_memcpy_success == hipSuccess);

    // clean up gpu memory after our calculations
    dev_a_cleanup_memset_success = hipMemset(dev_a, 0, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    dev_b_cleanup_memset_success = hipMemset(dev_b, 0, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    dev_c_cleanup_memset_success = hipMemset(dev_c, 0, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    assert(dev_a_cleanup_memset_success == hipSuccess);
    assert(dev_b_cleanup_memset_success == hipSuccess);
    assert(dev_c_cleanup_memset_success == hipSuccess);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
