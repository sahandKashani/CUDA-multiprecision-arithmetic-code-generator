#include "hip/hip_runtime.h"
#include "benchmarks.cuh"
#include "bignum_types.h"
#include "bignum_conversions.h"
#include "constants.h"
#include "operation_check.h"
#include "ptx_inline_operations.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), void (*checking_function)(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b), char* operation_name);

void addition_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b);
__global__ void addition_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);

void subtraction_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b);
__global__ void subtraction_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// BENCHMARKS /////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    addition_benchmark(host_c, host_a, host_b);
    subtraction_benchmark(host_c, host_a, host_b);
}

void addition_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    binary_operator_benchmark(host_c, host_a, host_b, addition_kernel, addition_check, "addition");
}

void subtraction_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    binary_operator_benchmark(host_c, host_a, host_b, subtraction_kernel, subtraction_check, "subtraction");
}

////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////// KERNELS ///////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

__global__ void addition_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    ptx_add(dev_c, dev_a, dev_b, blockIdx.x * blockDim.x + threadIdx.x);
}

__global__ void subtraction_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    ptx_sub(dev_c, dev_a, dev_b, blockIdx.x * blockDim.x + threadIdx.x);
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////// GENERIC LAUNCH CONFIGURATION ////////////////////////
////////////////////////////////////////////////////////////////////////////////

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), void (*checking_function)(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b), char* operation_name)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);
    assert(kernel != NULL);
    assert(checking_function != NULL);
    assert(operation_name != NULL);

    // arrange data in coalesced form
    bignum_array_to_coalesced_bignum_array(host_a);
    bignum_array_to_coalesced_bignum_array(host_b);
    bignum_array_to_coalesced_bignum_array(host_c);

    // device operands (dev_a, dev_b) and results (dev_c)
    uint32_t* dev_a;
    uint32_t* dev_b;
    uint32_t* dev_c;

    // allocate gpu memory
    hipError_t dev_a_malloc_success = hipMalloc((void**) &dev_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_b_malloc_success = hipMalloc((void**) &dev_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_c_malloc_success = hipMalloc((void**) &dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));

    assert(dev_a_malloc_success == hipSuccess);
    assert(dev_b_malloc_success == hipSuccess);
    assert(dev_c_malloc_success == hipSuccess);

    // copy operands to device memory
    hipError_t dev_a_memcpy_succes = hipMemcpy(dev_a, host_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipError_t dev_b_memcpy_succes = hipMemcpy(dev_b, host_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);

    assert(dev_a_memcpy_succes == hipSuccess);
    assert(dev_b_memcpy_succes == hipSuccess);

    // execute kernel
    printf("Performing \"%s\" on GPU ... ", operation_name);
    fflush(stdout);

    kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dev_c, dev_a, dev_b);

    printf("done\n");
    fflush(stdout);

    // copy results back to host
    hipError_t dev_c_memcpy_success = hipMemcpy(host_c, dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    assert(dev_c_memcpy_success == hipSuccess);

    // put data back to non-coalesced form
    coalesced_bignum_array_to_bignum_array(host_a);
    coalesced_bignum_array_to_bignum_array(host_b);
    coalesced_bignum_array_to_bignum_array(host_c);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // check if the results of the addition are correct by telling gmp to do
    // them on the cpu as a verification.
    checking_function(host_c, host_a, host_b);
}
