#include "hip/hip_runtime.h"
#include "benchmarks.cuh"
#include "bignum_types.h"
#include "bignum_conversions.h"
#include "constants.h"
#include "operation_check.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>

void benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    addition_benchmark(host_c, host_a, host_b);
    subtraction_benchmark(host_c, host_a, host_b);
    // modular_subtraction_benchmark(host_c, host_a, host_b);
}

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), void (*checking_function)(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b))
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);
    assert(kernel != NULL);
    assert(checking_function != NULL);

    // arrange data in coalesced form
    bignum_array_to_coalesced_bignum_array(host_a);
    bignum_array_to_coalesced_bignum_array(host_b);
    bignum_array_to_coalesced_bignum_array(host_c);

    // device operands (dev_a, dev_b) and results (dev_c)
    uint32_t* dev_a;
    uint32_t* dev_b;
    uint32_t* dev_c;

    // allocate gpu memory
    hipError_t dev_a_malloc_success = hipMalloc((void**) &dev_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_b_malloc_success = hipMalloc((void**) &dev_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_c_malloc_success = hipMalloc((void**) &dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));

    assert(dev_a_malloc_success == hipSuccess);
    assert(dev_b_malloc_success == hipSuccess);
    assert(dev_c_malloc_success == hipSuccess);

    // copy operands to device memory
    hipError_t dev_a_memcpy_succes = hipMemcpy(dev_a, host_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipError_t dev_b_memcpy_succes = hipMemcpy(dev_b, host_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);

    assert(dev_a_memcpy_succes == hipSuccess);
    assert(dev_b_memcpy_succes == hipSuccess);

    // execute kernel
    kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dev_c, dev_a, dev_b);

    // copy results back to host
    hipError_t dev_c_memcpy_success = hipMemcpy(host_c, dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    assert(dev_c_memcpy_success == hipSuccess);

    // put data back to non-coalesced form
    coalesced_bignum_array_to_bignum_array(host_a);
    coalesced_bignum_array_to_bignum_array(host_b);
    coalesced_bignum_array_to_bignum_array(host_c);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // check if the results of the addition are correct by telling gmp to do
    // them on the cpu as a verification.
    checking_function(host_c, host_a, host_b);
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// ADDITION ///////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void addition_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    binary_operator_benchmark(host_c, host_a, host_b, addition_kernel, addition_check);
}

__global__ void addition_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    addition(dev_c, dev_a, dev_b);
}

__device__ void addition(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    while (tid < NUMBER_OF_BIGNUMS)
    {
        asm("add.cc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_a[COAL_IDX(0, tid)]),
              "r" (dev_b[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("addc.cc.u32 %0, %1, %2;"
                : "=r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_a[COAL_IDX(i, tid)]),
                  "r" (dev_b[COAL_IDX(i, tid)]));
        }

        asm("addc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
              "r" (dev_b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        tid += stride;
    }
}

////////////////////////////////////////////////////////////////////////////////
///////////////////////////////// SUBTRACTION //////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void subtraction_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    binary_operator_benchmark(host_c, host_a, host_b, subtraction_kernel, subtraction_check);
}

__global__ void subtraction_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    subtraction(dev_c, dev_a, dev_b);
}

__device__ void subtraction(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    while (tid < NUMBER_OF_BIGNUMS)
    {
        asm("sub.cc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_a[COAL_IDX(0, tid)]),
              "r" (dev_b[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("subc.cc.u32 %0, %1, %2;"
                : "=r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_a[COAL_IDX(i, tid)]),
                  "r" (dev_b[COAL_IDX(i, tid)]));
        }

        asm("subc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
              "r" (dev_b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        tid += stride;
    }
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////// MODULAR ADDITION ///////////////////////////////
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
////////////////////////////// MODULAR SUBTRACTION /////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void modular_subtraction_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    binary_operator_benchmark(host_c, host_a, host_b, modular_subtraction_kernel, subtraction_check);
}

__global__ void modular_subtraction_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    modular_subtraction(dev_c, dev_a, dev_b);
}

__device__ void modular_subtraction(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    while (tid < NUMBER_OF_BIGNUMS)
    {
        asm("sub.cc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_a[COAL_IDX(0, tid)]),
              "r" (dev_b[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("subc.cc.u32 %0, %1, %2;"
                : "=r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_a[COAL_IDX(i, tid)]),
                  "r" (dev_b[COAL_IDX(i, tid)]));
        }

        asm("subc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
              "r" (dev_b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        tid += stride;
    }
}

////////////////////////////////////////////////////////////////////////////////
///////////////////////////////// MULTIPLICATION ///////////////////////////////
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
///////////////////////////// MODULAR MULTIPLICATION ///////////////////////////
////////////////////////////////////////////////////////////////////////////////
