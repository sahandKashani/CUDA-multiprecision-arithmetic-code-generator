#include "hip/hip_runtime.h"
#include "benchmarks.cuh"
#include "bignum_types.h"
#include "bignum_conversions.h"
#include "constants.h"
#include "operation_check.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), void (*checking_function)(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b), char* operation_name);

void addition_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b);
__global__ void addition_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__device__ void addition(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);

void subtraction_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b);
__global__ void subtraction_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__device__ void subtraction(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// BENCHMARKS /////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    addition_benchmark(host_c, host_a, host_b);
    subtraction_benchmark(host_c, host_a, host_b);
}

void addition_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    binary_operator_benchmark(host_c, host_a, host_b, addition_kernel, addition_check, "addition");
}

void subtraction_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    binary_operator_benchmark(host_c, host_a, host_b, subtraction_kernel, subtraction_check, "subtraction");
}

////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////// KERNELS ///////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

__global__ void addition_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    addition(dev_c, dev_a, dev_b);
}

__global__ void subtraction_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    subtraction(dev_c, dev_a, dev_b);
}

////////////////////////////////////////////////////////////////////////////////
//////////////////////////////// DEVICE FUNCTIONS //////////////////////////////
////////////////////////////////////////////////////////////////////////////////

__device__ void addition(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    while (tid < NUMBER_OF_BIGNUMS)
    {
        asm("add.cc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_a[COAL_IDX(0, tid)]),
              "r" (dev_b[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("addc.cc.u32 %0, %1, %2;"
                : "=r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_a[COAL_IDX(i, tid)]),
                  "r" (dev_b[COAL_IDX(i, tid)]));
        }

        asm("addc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
              "r" (dev_b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        tid += stride;
    }
}

__device__ void subtraction(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    while (tid < NUMBER_OF_BIGNUMS)
    {
        asm("sub.cc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_a[COAL_IDX(0, tid)]),
              "r" (dev_b[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("subc.cc.u32 %0, %1, %2;"
                : "=r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_a[COAL_IDX(i, tid)]),
                  "r" (dev_b[COAL_IDX(i, tid)]));
        }

        asm("subc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
              "r" (dev_b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        tid += stride;
    }
}

__device__ void modular_addition(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b, uint32_t* dev_m)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    while (tid < NUMBER_OF_BIGNUMS)
    {
        // addition
        asm("add.cc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_a[COAL_IDX(0, tid)]),
              "r" (dev_b[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("addc.cc.u32 %0, %1, %2;"
                : "=r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_a[COAL_IDX(i, tid)]),
                  "r" (dev_b[COAL_IDX(i, tid)]));
        }

        asm("addc.u32 %0, %1, %2;"
            : "=r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_a[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
              "r" (dev_b[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        // subtraction
        asm("sub.cc.u32 %0, %0, %1;"
            : "+r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_m[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("subc.cc.u32 %0, %0, %1;"
                : "+r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_m[COAL_IDX(i, tid)]));
        }

        // we want the borrow bit (unlike for normal subtraction)
        asm("subc.cc.u32 %0, %0, %1;"
            : "+r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_m[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        uint32_t borrow[BIGNUM_NUMBER_OF_WORDS];
        asm("subc.u32 %0, 0, ");
        for

        uint32_t mask[BIGNUM_NUMBER_OF_WORDS];
        for (uint32_t i = 0; i < BIGNUM_NUMBER_OF_WORDS; i++)
        {
            mask[i] = 0;
        }

        // mask = 0 until now
        // now do mask = mask - mask

        // subtraction
        asm("subc.cc.u32 %0, %0, %1;"
            : "+r"(dev_c[COAL_IDX(0, tid)])
            : "r" (dev_m[COAL_IDX(0, tid)]));

        #pragma unroll
        for (uint32_t i = 1; i < BIGNUM_NUMBER_OF_WORDS - 1; i++)
        {
            asm("subc.cc.u32 %0, %0, %1;"
                : "+r"(dev_c[COAL_IDX(i, tid)])
                : "r" (dev_m[COAL_IDX(i, tid)]));
        }

        // we want the borrow bit (unlike for normal subtraction)
        asm("subc.cc.u32 %0, %0, %1;"
            : "+r"(dev_c[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)])
            : "r" (dev_m[COAL_IDX(BIGNUM_NUMBER_OF_WORDS - 1, tid)]));

        tid += stride;
    }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////// GENERIC LAUNCH CONFIGURATION ////////////////////////
////////////////////////////////////////////////////////////////////////////////

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), void (*checking_function)(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b), char* operation_name)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);
    assert(kernel != NULL);
    assert(checking_function != NULL);
    assert(operation_name != NULL);

    // arrange data in coalesced form
    bignum_array_to_coalesced_bignum_array(host_a);
    bignum_array_to_coalesced_bignum_array(host_b);
    bignum_array_to_coalesced_bignum_array(host_c);

    // device operands (dev_a, dev_b) and results (dev_c)
    uint32_t* dev_a;
    uint32_t* dev_b;
    uint32_t* dev_c;

    // allocate gpu memory
    hipError_t dev_a_malloc_success = hipMalloc((void**) &dev_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_b_malloc_success = hipMalloc((void**) &dev_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_c_malloc_success = hipMalloc((void**) &dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));

    assert(dev_a_malloc_success == hipSuccess);
    assert(dev_b_malloc_success == hipSuccess);
    assert(dev_c_malloc_success == hipSuccess);

    // copy operands to device memory
    hipError_t dev_a_memcpy_succes = hipMemcpy(dev_a, host_a, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipError_t dev_b_memcpy_succes = hipMemcpy(dev_b, host_b, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);

    assert(dev_a_memcpy_succes == hipSuccess);
    assert(dev_b_memcpy_succes == hipSuccess);

    // execute kernel
    printf("Performing \"%s\" on GPU ... ", operation_name);
    fflush(stdout);

    kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dev_c, dev_a, dev_b);

    printf("done\n");
    fflush(stdout);

    // copy results back to host
    hipError_t dev_c_memcpy_success = hipMemcpy(host_c, dev_c, NUMBER_OF_BIGNUMS * BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    assert(dev_c_memcpy_success == hipSuccess);

    // put data back to non-coalesced form
    coalesced_bignum_array_to_bignum_array(host_a);
    coalesced_bignum_array_to_bignum_array(host_b);
    coalesced_bignum_array_to_bignum_array(host_c);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // check if the results of the addition are correct by telling gmp to do
    // them on the cpu as a verification.
    checking_function(host_c, host_a, host_b);
}
