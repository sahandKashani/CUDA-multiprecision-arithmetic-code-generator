#include "hip/hip_runtime.h"
#include "benchmarks.cuh"
#include "bignum_types.h"
#include "bignum_conversions.h"
#include "constants.h"
#include "operation_check.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), void (*checking_function)(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b), char* operation_name);

void add_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b);
void sub_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b);
void mul_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b);

__global__ void add_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__global__ void sub_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);
__global__ void mul_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b);

__device__ void add(uint32_t* c, uint32_t* a, uint32_t* b, uint32_t tid);
__device__ void sub(uint32_t* c, uint32_t* a, uint32_t* b, uint32_t tid);
__device__ void mul(uint32_t* c, uint32_t* a, uint32_t* b, uint32_t tid);

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// BENCHMARKS /////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);

    add_benchmark(host_c, host_a, host_b);
    sub_benchmark(host_c, host_a, host_b);
    mul_benchmark(host_c, host_a, host_b);
}

void add_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    binary_operator_benchmark(host_c, host_a, host_b, add_kernel, add_check, "addition");
}

void sub_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    binary_operator_benchmark(host_c, host_a, host_b, sub_kernel, sub_check, "subtraction");
}

void mul_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b)
{
    binary_operator_benchmark(host_c, host_a, host_b, mul_kernel, mul_check, "multiplication");
}

////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////// KERNELS ///////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

__global__ void add_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    add(dev_c, dev_a, dev_b, tid);
}

__global__ void sub_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    sub(dev_c, dev_a, dev_b, tid);
}

__global__ void mul_kernel(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    mul(dev_c, dev_a, dev_b, tid);
}

////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// OPERATIONS /////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

__device__ void add(uint32_t* c_glo, uint32_t* a_glo, uint32_t* b_glo, uint32_t tid)
{
    asm("add.cc.u32 %0, %1, %2;"
        : "=r"(c_glo[COAL_IDX(0, tid)])
        : "r" (a_glo[COAL_IDX(0, tid)]),
          "r" (b_glo[COAL_IDX(0, tid)]));

    #pragma unroll
    for (uint32_t i = 1; i < MAX_BIGNUM_NUMBER_OF_WORDS - 1; i++)
    {
        asm("addc.cc.u32 %0, %1, %2;"
            : "=r"(c_glo[COAL_IDX(i, tid)])
            : "r" (a_glo[COAL_IDX(i, tid)]),
              "r" (b_glo[COAL_IDX(i, tid)]));
    }

    asm("addc.u32 %0, %1, %2;"
        : "=r"(c_glo[COAL_IDX(MAX_BIGNUM_NUMBER_OF_WORDS - 1, tid)])
        : "r" (a_glo[COAL_IDX(MAX_BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
          "r" (b_glo[COAL_IDX(MAX_BIGNUM_NUMBER_OF_WORDS - 1, tid)]));
}

__device__ void sub(uint32_t* c_glo, uint32_t* a_glo, uint32_t* b_glo, uint32_t tid)
{
    asm("sub.cc.u32 %0, %1, %2;"
        : "=r"(c_glo[COAL_IDX(0, tid)])
        : "r" (a_glo[COAL_IDX(0, tid)]),
          "r" (b_glo[COAL_IDX(0, tid)]));

    #pragma unroll
    for (uint32_t i = 1; i < MAX_BIGNUM_NUMBER_OF_WORDS - 1; i++)
    {
        asm("subc.cc.u32 %0, %1, %2;"
            : "=r"(c_glo[COAL_IDX(i, tid)])
            : "r" (a_glo[COAL_IDX(i, tid)]),
              "r" (b_glo[COAL_IDX(i, tid)]));
    }

    asm("subc.u32 %0, %1, %2;"
        : "=r"(c_glo[COAL_IDX(MAX_BIGNUM_NUMBER_OF_WORDS - 1, tid)])
        : "r" (a_glo[COAL_IDX(MAX_BIGNUM_NUMBER_OF_WORDS - 1, tid)]),
          "r" (b_glo[COAL_IDX(MAX_BIGNUM_NUMBER_OF_WORDS - 1, tid)]));
}

__device__ void mul(uint32_t* c_glo, uint32_t* a_glo, uint32_t* b_glo, uint32_t tid)
{
    // ATTENTION: Assuming "a" and "b" are n-bit bignums, their multiplication
    // can give a bignum of length 2n-bits. Since we are coding a generic
    // multiplication, we will use this information to do less loops, so we use
    // MIN_BIGNUM_NUMBER_OF_WORDS to represent "a" and "b", and
    // MAX_BIGNUM_NUMBER_OF_WORDS to represent "c".
    uint32_t a_loc[MIN_BIGNUM_NUMBER_OF_WORDS];
    uint32_t b_loc[MIN_BIGNUM_NUMBER_OF_WORDS];
    uint32_t c_loc[MAX_BIGNUM_NUMBER_OF_WORDS];
    uint32_t carry[MAX_BIGNUM_NUMBER_OF_WORDS];

    // Example of the schoolbook multiplication algorithm we will use:
    //
    //                                      A[4]   A[3]   A[2]   A[1]   A[0]
    //                                    * B[4]   B[3]   B[2]   B[1]   B[0]
    // -----------------------------------------------------------------------
    // |      |      |      |      |      |      |      |      | B[0] * A[0] |
    // |      |      |      |      |      |      |      | B[0] * A[1] |      |
    // |      |      |      |      |      |      | B[0] * A[2] |      |      |
    // |      |      |      |      |      | B[0] * A[3] |      |      |      |
    // |      |      |      |      | B[0] * A[4] |      |      |      |      |
    // |      |      |      |      |      |      |      | B[1] * A[0] |      |
    // |      |      |      |      |      |      | B[1] * A[1] |      |      |
    // |      |      |      |      |      | B[1] * A[2] |      |      |      |
    // |      |      |      |      | B[1] * A[3] |      |      |      |      |
    // |      |      |      | B[1] * A[4] |      |      |      |      |      |
    // |      |      |      |      |      |      | B[2] * A[0] |      |      |
    // |      |      |      |      |      | B[2] * A[1] |      |      |      |
    // |      |      |      |      | B[2] * A[2] |      |      |      |      |
    // |      |      |      | B[2] * A[3] |      |      |      |      |      |
    // |      |      | B[2] * A[4] |      |      |      |      |      |      |
    // |      |      |      |      |      | B[3] * A[0] |      |      |      |
    // |      |      |      |      | B[3] * A[1] |      |      |      |      |
    // |      |      |      | B[3] * A[2] |      |      |      |      |      |
    // |      |      | B[3] * A[3] |      |      |      |      |      |      |
    // |      | B[3] * A[4] |      |      |      |      |      |      |      |
    // |      |      |      |      | B[4] * A[0] |      |      |      |      |
    // |      |      |      | B[4] * A[1] |      |      |      |      |      |
    // |      |      | B[4] * A[2] |      |      |      |      |      |      |
    // |      | B[4] * A[3] |      |      |      |      |      |      |      |
    // + B[4] * A[4] |      |      |      |      |      |      |      |      |
    // -----------------------------------------------------------------------
    // | C[9] | C[8] | C[7] | C[6] | C[5] | C[4] | C[3] | C[2] | C[1] | C[0] |

    // Because of CUDA carry propagation problems (the carry flag is only kept
    // for the next assembly instruction), we will have to order the steps in
    // the following way:
    //
    //                                      A[4]   A[3]   A[2]   A[1]   A[0]
    //                                    * B[4]   B[3]   B[2]   B[1]   B[0]
    // -----------------------------------------------------------------------
    // |      |      |      |      |      |      |      |      | B[0] * A[0] |
    // |      |      |      |      |      |      |      | B[0] * A[1] |      |
    // |      |      |      |      |      |      |      | B[1] * A[0] |      |
    // |      |      |      |      |      |      | B[0] * A[2] |      |      |
    // |      |      |      |      |      |      | B[1] * A[1] |      |      |
    // |      |      |      |      |      |      | B[2] * A[0] |      |      |
    // |      |      |      |      |      | B[0] * A[3] |      |      |      |
    // |      |      |      |      |      | B[1] * A[2] |      |      |      |
    // |      |      |      |      |      | B[2] * A[1] |      |      |      |
    // |      |      |      |      |      | B[3] * A[0] |      |      |      |
    // |      |      |      |      | B[0] * A[4] |      |      |      |      |
    // |      |      |      |      | B[1] * A[3] |      |      |      |      |
    // |      |      |      |      | B[2] * A[2] |      |      |      |      |
    // |      |      |      |      | B[3] * A[1] |      |      |      |      |
    // |      |      |      |      | B[4] * A[0] |      |      |      |      |
    // |      |      |      | B[1] * A[4] |      |      |      |      |      |
    // |      |      |      | B[2] * A[3] |      |      |      |      |      |
    // |      |      |      | B[3] * A[2] |      |      |      |      |      |
    // |      |      |      | B[4] * A[1] |      |      |      |      |      |
    // |      |      | B[2] * A[4] |      |      |      |      |      |      |
    // |      |      | B[3] * A[3] |      |      |      |      |      |      |
    // |      |      | B[4] * A[2] |      |      |      |      |      |      |
    // |      | B[3] * A[4] |      |      |      |      |      |      |      |
    // |      | B[4] * A[3] |      |      |      |      |      |      |      |
    // + B[4] * A[4] |      |      |      |      |      |      |      |      |
    // -----------------------------------------------------------------------
    // | C[9] | C[8] | C[7] | C[6] | C[5] | C[4] | C[3] | C[2] | C[1] | C[0] |

    // A = operand 1, B = operand 2, D = carry, C = result
    //
    // mad.lo.u32    C[0], B[0], A[0], 0
    //
    // mad.hi.u32    C[1], B[0], A[1], 0
    // mad.lo.cc.u32 C[1], B[0], A[1], C[1]
    // addc.u32      D[2], D[2], 0
    // mad.lo.cc.u32 C[1], B[1], A[0], C[1]
    // addc.u32      D[2], D[2], 0
    //
    // mad.hi.u32    C[2], B[0], A[1], 0
    // mad.hi.u32    j


    // extended-precision multiply: [C[3],C[2],C[1],C[0]] = [A[1],A[0]] * [B[1],B[0]]
    // mul.lo.u32     C[0], B[0], A[0]      ; // C[0]  = (A[0]*B[0]).[31:0]             , no  carry-out
    // mul.hi.u32     C[1], B[0], A[0]      ; // C[1]  = (A[0]*B[0]).[63:32]            , no  carry-out
    // mad.lo.cc.u32  C[1], B[0], A[1], C[1]; // C[1] += (A[1]*B[0]).[31:0]             , may carry-out
    // madc.hi.u32    C[2], B[0], A[1], 0   ; // C[2]  = (A[1]*B[0]).[63:32] + carry-in , no  carry-out
    // mad.lo.cc.u32  C[1], B[1], A[0], C[1]; // C[1] += (A[0]*B[1]).[31:0]             , may carry-out
    // madc.hi.cc.u32 C[2], B[1], A[0], C[2]; // C[2] += (A[0]*B[1]).[63:32] + carry-in , may carry-out
    // addc.u32       C[3], 0   , 0         ; // C[3]  = carry-in                       , no  carry-out
    // mad.lo.cc.u32  C[2], B[1], A[1], C[2]; // C[2] += (A[1]*B[1]).[31:0]             , may carry-out
    // madc.hi.u32    C[3], B[1], A[1], C[3]; // C[3] += (A[1]*B[1]).[63:32] + carry-in
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////// GENERIC LAUNCH CONFIGURATION ////////////////////////
////////////////////////////////////////////////////////////////////////////////

void binary_operator_benchmark(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b, void (*kernel)(uint32_t* dev_c, uint32_t* dev_a, uint32_t* dev_b), void (*checking_function)(uint32_t* host_c, uint32_t* host_a, uint32_t* host_b), char* operation_name)
{
    assert(host_a != NULL);
    assert(host_b != NULL);
    assert(host_c != NULL);
    assert(kernel != NULL);
    assert(checking_function != NULL);
    assert(operation_name != NULL);

    // arrange data in coalesced form
    bignum_array_to_coalesced_bignum_array(host_a);
    bignum_array_to_coalesced_bignum_array(host_b);
    bignum_array_to_coalesced_bignum_array(host_c);

    // device operands (dev_a, dev_b) and results (dev_c)
    uint32_t* dev_a;
    uint32_t* dev_b;
    uint32_t* dev_c;

    // allocate gpu memory
    hipError_t dev_a_malloc_success = hipMalloc((void**) &dev_a, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_b_malloc_success = hipMalloc((void**) &dev_b, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));
    hipError_t dev_c_malloc_success = hipMalloc((void**) &dev_c, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t));

    assert(dev_a_malloc_success == hipSuccess);
    assert(dev_b_malloc_success == hipSuccess);
    assert(dev_c_malloc_success == hipSuccess);

    // copy operands to device memory
    hipError_t dev_a_memcpy_succes = hipMemcpy(dev_a, host_a, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipError_t dev_b_memcpy_succes = hipMemcpy(dev_b, host_b, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyHostToDevice);

    assert(dev_a_memcpy_succes == hipSuccess);
    assert(dev_b_memcpy_succes == hipSuccess);

    // execute kernel
    printf("Performing \"%s\" on GPU ... ", operation_name);
    fflush(stdout);

    kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(dev_c, dev_a, dev_b);

    printf("done\n");
    fflush(stdout);

    // copy results back to host
    hipError_t dev_c_memcpy_success = hipMemcpy(host_c, dev_c, NUMBER_OF_BIGNUMS * MAX_BIGNUM_NUMBER_OF_WORDS * sizeof(uint32_t), hipMemcpyDeviceToHost);

    assert(dev_c_memcpy_success == hipSuccess);

    // put data back to non-coalesced form
    coalesced_bignum_array_to_bignum_array(host_a);
    coalesced_bignum_array_to_bignum_array(host_b);
    coalesced_bignum_array_to_bignum_array(host_c);

    // free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // check if the results of the addition are correct by telling gmp to do
    // them on the cpu as a verification.
    checking_function(host_c, host_a, host_b);
}
