#include "hip/hip_runtime.h"
#include "conversions.h"
#include <stdio.h>
#include <stdlib.h>
#include <gmp.h>

// __global__ void test_kernel(int* dev_c, int a, int b);
char* generate_random_number(unsigned int index, unsigned int seed,
                             unsigned int bits, unsigned int base);

int main(void)
{
    printf("Testing inline PTX\n");

    int i = 0;
    char* number_str;

    number_str = generate_random_number(i++, SEED, RANDOM_NUMBER_BIT_RANGE, BASE);

    bignum a;
    string_to_bignum(number_str, a);
    free(number_str);

    // hipMalloc((void**) &dev_c, sizeof(int));
    // test_kernel<<<1, 1>>>(dev_c, a, b);
    // hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    // printf("%d + %d = %d\n", a, b, c);
    // hipFree(dev_c);
}

/**
 * Generates the i'th random number from the seed, where "i" is the "index"
 * value passed as a parameter. Remember to call free() on the returned string
 * once you don't need it anymore.
 * @param  index "Index" of the random number.
 * @param  seed  Seed of the random number generator.
 * @param  bits  Bit precision requested.
 * @param  base  Base of the number returned in the string (2 until 62)
 * @return       String representing the binary version of the number.
 */
char* generate_random_number(unsigned int index, unsigned int seed,
                             unsigned int bits, unsigned int base)
{
    // random number generator initialization
    gmp_randstate_t random_state;
    gmp_randinit_default(random_state);
    // incorporated seed in generator
    gmp_randseed_ui(random_state, seed);

    // initialize test vector operands and result
    mpz_t number;
    mpz_init(number);

    // generate random number
    mpz_urandomb(number, random_state, bits);
    for (int i = 0; i < index; i++)
    {
        mpz_urandomb(number, random_state, bits);
    }

    // get binary string version
    char* str_number = mpz_get_str(NULL, base, number);
    pad_string_with_zeros(&str_number);

    // get memory back from operands and results
    mpz_clear(number);

    // get memory back from gmp_randstate_t
    gmp_randclear(random_state);

    return str_number;
}

// __global__ void test_kernel(int* dev_c, int a, int b)
// {
//     int c;

//     asm("{"
//         "    add.u32 %0, %1, %2;"
//         "}"
//         : "=r"(c) : "r"(a), "r"(b)
//         );

//     *dev_c = c;
// }
